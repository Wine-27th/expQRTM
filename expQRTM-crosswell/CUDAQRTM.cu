#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <string.h>
#include <hip/hip_runtime.h>
#define BLOCK_SIZE 16		// we assume block size equals 16
#define PI 3.1415926
#include "Myfunctions.h"
using namespace std;

// define multistream to prepare for streaming execution
struct Multistream
{
	hipStream_t stream,stream_back;
};


//==========================================================
//  This subroutine is used for initializating wavefield variables
//  =========================================================
__global__ void cuda_kernel_initialization
(
	int ntx, int ntz, hipfftComplex *u0, hipfftComplex *u1, hipfftComplex *u2, 
	hipfftComplex *uk0, hipfftComplex *uk, hipfftComplex *Lap, hipfftComplex *amp_Lap, hipfftComplex *pha_Lap, hipfftComplex *sta_Lap
)
{
	int bx=blockIdx.x;
	int by=blockIdx.y;
	int tx=threadIdx.x;
	int ty=threadIdx.y;
	int iz=by*BLOCK_SIZE+ty;
	int ix=bx*BLOCK_SIZE+tx;
	int ip=iz*ntx+ix;	

	if(iz>=0&&iz<=ntz-1&&ix>=0&&ix<=ntx-1)
	{
		u0[ip].x=0.0; u0[ip].y=0.0;
		u1[ip].x=0.0; u1[ip].y=0.0;
		u2[ip].x=0.0; u2[ip].y=0.0;
		uk0[ip].x=0.0; uk0[ip].y=0.0; 
		uk[ip].x=0.0; uk[ip].y=0.0; 
		Lap[ip].x=0.0; Lap[ip].y=0.0; 
		amp_Lap[ip].x=0.0; amp_Lap[ip].y=0.0; 
		pha_Lap[ip].x=0.0; pha_Lap[ip].y=0.0; 
		sta_Lap[ip].x=0.0; sta_Lap[ip].y=0.0;
	}
	__syncthreads();	
}


//==========================================================
//  This subroutine is used for updating wavefield variables
//  =========================================================
__global__ void cuda_kernel_update
(
	int ntx, int ntz, hipfftComplex *u0, hipfftComplex *u1, hipfftComplex *u2
)
{
	int bx=blockIdx.x;
	int by=blockIdx.y;
	int tx=threadIdx.x;
	int ty=threadIdx.y;
	int iz=by*BLOCK_SIZE+ty;
	int ix=bx*BLOCK_SIZE+tx;
	int ip=iz*ntx+ix;	

	if(iz>=0&&iz<=ntz-1&&ix>=0&&ix<=ntx-1)
	{
		u0[ip].x=u1[ip].x;
		u0[ip].y=u1[ip].y;
		u1[ip].x=u2[ip].x;
		u1[ip].y=u2[ip].y;
	}
	__syncthreads();
}


//==========================================================
//  This subroutine is used for initializating image variables
//  =========================================================
__global__ void cuda_kernel_initialization_images
(
	int ntx, int ntz, float *image_cor, float *image_nor, float *image_sources, float *image_receivers
)
{
	int bx=blockIdx.x;
	int by=blockIdx.y;
	int tx=threadIdx.x;
	int ty=threadIdx.y;
	int iz=by*BLOCK_SIZE+ty;
	int ix=bx*BLOCK_SIZE+tx;
	int ip=iz*ntx+ix;	

	if(iz>=0&&iz<=ntz-1&&ix>=0&&ix<=ntx-1)
	{
		image_cor[ip]=0;
		image_nor[ip]=0;
		image_sources[ip]=0;
		image_receivers[ip]=0;
	}

	__syncthreads();
}


//==========================================================
//  This subroutine is used for defining k
// =========================================================
__global__ void cuda_kernel_k_define
(
	int ntx, int ntz, float dx, float dz, float *kx, float *kz
)
{
	int bx=blockIdx.x;
	int by=blockIdx.y;
	int tx=threadIdx.x;
	int ty=threadIdx.y;
	int iz=by*BLOCK_SIZE+ty;
	int ix=bx*BLOCK_SIZE+tx;

	int nxh=ntx/2;
	int nzh=ntz/2;
	float dkx=1.0/(ntx*dx);
	float dkz=1.0/(ntz*dz);

	if(ix>=0&&ix<=nxh)
	{
		kx[ix]=2*PI*ix*dkx;
	}
	if(ix>nxh&&ix<ntx)
	{
		kx[ix]=kx[ntx-ix];
	}
	if(iz>=0&&iz<=nzh)
	{
		kz[iz]=2*PI*iz*dkz;
	}
	if(iz>nzh&&iz<ntz)
	{
		kz[iz]=kz[ntz-iz];
	}
	__syncthreads();
}


//==========================================================
//  This subroutine is used for calculating forward wavefileds in k-space
//  ========================================================
__global__ void cuda_kernel_visco_PSM_2d_forward_k_space
(
	float beta1, float beta2,
	int it, int nt, int ntx, int ntz, float dx, float dz, float dt, 
	float *vp, float *Gamma, float averGamma, float f0, float Omega0, float alphaorder,
	float *kx, float *kz, 
	hipfftComplex *uk, hipfftComplex *uk0, 
	hipfftComplex *Lap_uk, hipfftComplex *amp_uk, hipfftComplex *pha_uk, hipfftComplex *sta_uk
)
{
	int bx=blockIdx.x;
	int by=blockIdx.y;
	int tx=threadIdx.x;
	int ty=threadIdx.y;
	int iz=by*BLOCK_SIZE+ty;
	int ix=bx*BLOCK_SIZE+tx;
	int ip=iz*ntx+ix;

	if(iz>=0&&iz<=ntz-1&&ix>=0&&ix<=ntx-1)
	{
		Lap_uk[ip].x=-(powf(kx[ix],2)+powf(kz[iz],2))*uk[ip].x;
		Lap_uk[ip].y=-(powf(kx[ix],2)+powf(kz[iz],2))*uk[ip].y;
		if(beta1!=0)
		{
			pha_uk[ip].x=powf((powf(kx[ix],2)+powf(kz[iz],2)), averGamma+1)*uk[ip].x;
			pha_uk[ip].y=powf((powf(kx[ix],2)+powf(kz[iz],2)), averGamma+1)*uk[ip].y;
		}
		if(beta2!=0)
		{
			amp_uk[ip].x=powf((powf(kx[ix],2)+powf(kz[iz],2)), averGamma+0.5)*(uk[ip].x-uk0[ip].x)/dt;
			amp_uk[ip].y=powf((powf(kx[ix],2)+powf(kz[iz],2)), averGamma+0.5)*(uk[ip].y-uk0[ip].y)/dt;
		}	
		if(beta2<0)
		{
			sta_uk[ip].x=powf((powf(kx[ix],2)+powf(kz[iz],2)), 0.5*alphaorder)*(uk[ip].x-uk0[ip].x)/dt;
			sta_uk[ip].y=powf((powf(kx[ix],2)+powf(kz[iz],2)), 0.5*alphaorder)*(uk[ip].y-uk0[ip].y)/dt;
		}						
		uk0[ip].x=uk[ip].x;
		uk0[ip].y=uk[ip].y;
	}
	__syncthreads();
}


//==========================================================
//  This subroutine is used for calculating forward wavefileds in x-space
//  ========================================================
__global__ void cuda_kernel_visco_PSM_2d_forward_x_space
(
	float beta1, float beta2,
	int it, int nt, int ntx, int ntz, int nx, int nz, int L, float dx, float dz, float dt, 
	float *vp, float *Gamma, float averGamma, float f0, float Omega0, float sigmafactor,
	float *seismogram, int *r_ix, int *r_iz, int rnmax, float *ricker, int s_ix, int s_iz,
	hipfftComplex *u0, hipfftComplex *u1, hipfftComplex *u2,
	hipfftComplex *Lap, hipfftComplex *amp_Lap, hipfftComplex *pha_Lap, hipfftComplex *sta_Lap,
	float *borders_up, float *borders_bottom, float *borders_left, float *borders_right,
	float *u2_final0, float *u2_final1,
	int Sto_Rec, int vp_type
)
{
	int bx=blockIdx.x;
	int by=blockIdx.y;
	int tx=threadIdx.x;
	int ty=threadIdx.y;
	int iz=by*BLOCK_SIZE+ty;
	int ix=bx*BLOCK_SIZE+tx;
	int ip=iz*ntx+ix;

	int icp;
	float eta, tau;

	if(iz>=0&&iz<=ntz-1&&ix>=0&&ix<=ntx-1)
	{		
		eta= -powf(vp[ip],2*Gamma[ip])*powf(Omega0,-2*Gamma[ip])*cos(Gamma[ip]*PI);
		tau= -powf(vp[ip],2*Gamma[ip]-1)*powf(Omega0,-2*Gamma[ip])*sin(Gamma[ip]*PI);

		// scale fft by dividing (ntx*ntz)
		Lap[ip].x=Lap[ip].x/(ntx*ntz);
		pha_Lap[ip].x=pha_Lap[ip].x/(ntx*ntz);
		amp_Lap[ip].x=amp_Lap[ip].x/(ntx*ntz);
		sta_Lap[ip].x=sta_Lap[ip].x/(ntx*ntz);

		u2[ip].x=powf(vp[ip]*cos(Gamma[ip]*PI/2),2)*powf(dt,2)
			*(
				Lap[ip].x
				+beta1*(eta*pha_Lap[ip].x-Lap[ip].x)
				+beta2*tau*amp_Lap[ip].x
				+tau*sigmafactor*sta_Lap[ip].x //-2*sigmafactor/powf(vp[ip]*cos(Gamma[ip]*PI/2),2)*sta_Lap[ip].x
			)
			+2*u1[ip].x-u0[ip].x;
	}
	// add Ricker source
	if(iz==s_iz&&ix==s_ix)
	{
		u2[ip].x+=ricker[it];
	}	
	// record Seismogram
	if(ix>=0&&ix<=rnmax-1)
	{
		seismogram[it*rnmax+ix] = u2[r_iz[ix]*ntx + r_ix[ix]].x;
	}

	// store borders and final two-step wavefileds for wavefield reconstruction
	if(Sto_Rec==0&&vp_type==2)
	{
		if(ix>=L&&ix<=ntx-L-1&&iz==L)
		{
			borders_up[it*nx+ix-L]=u2[ip].x;
		}
		if(ix>=L&&ix<=ntx-L-1&&iz==ntz-L-1)
		{
			borders_bottom[it*nx+ix-L]=u2[ip].x;
		}
		if(iz>=L&&iz<=ntz-L-1&&ix==L)
		{
			borders_left[it*nz+iz-L]=u2[ip].x;
		}
		if(iz>=L&&iz<=ntz-L-1&&ix==ntx-L-1)
		{
			borders_right[it*nz+iz-L]=u2[ip].x;
		}
		if(it==nt-1)
		{
			if(iz>=0&&iz<=ntz-1&&ix>=0&&ix<=ntx-1)
			{				
				u2_final0[ip]=u2[ip].x;
				u2_final1[ip]=u1[ip].x;				
			}			
		}
	}
	__syncthreads();
}


//==========================================================
//  This subroutine is used for writing checkpoints
//  ========================================================
__global__ void cuda_kernel_checkpoints_Out
(
	int it, int nt, int ntx, int ntz, int nx, int nz, int L, float dx, float dz, float dt, 
	hipfftComplex *u1, hipfftComplex *u2,
	float *u_cp, int N_cp, int *t_cp
)
{

	int bx=blockIdx.x;
	int by=blockIdx.y;
	int tx=threadIdx.x;
	int ty=threadIdx.y;
	int iz=by*BLOCK_SIZE+ty;
	int ix=bx*BLOCK_SIZE+tx;
	int ip=iz*ntx+ix;

	int icp;

	for(icp=0;icp<N_cp;icp++)
	{
		if(icp%2==1&&it==t_cp[icp])
		{
			if(iz>=0&&iz<=ntz-1&&ix>=0&&ix<=ntx-1)
			{
				u_cp[icp*ntx*ntz+ip]=u2[ip].x;
				u_cp[(icp-1)*ntx*ntz+ip]=u1[ip].x;
			}
		}
	}
	__syncthreads();
}


//==========================================================
//  This two subroutines are used for initializing Final two wavefileds
//  =========================================================
__global__ void cuda_kernel_initialization_Finals
(
	int ntx, int ntz, hipfftComplex *u0, hipfftComplex *u1, float *u2_final0, float *u2_final1
)
{
	int bx=blockIdx.x;
	int by=blockIdx.y;
	int tx=threadIdx.x;
	int ty=threadIdx.y;
	int iz=by*BLOCK_SIZE+ty;
	int ix=bx*BLOCK_SIZE+tx;
	int ip=iz*ntx+ix;	
	if(iz>=0&&iz<=ntz-1&&ix>=0&&ix<=ntx-1)
	{
		u0[ip].x=u2_final0[ip];
		u1[ip].x=u2_final1[ip];
	}
	__syncthreads();	
}


/*==========================================================
  This subroutine is used for calculating reconstructed wavefileds in k-space
  ===========================================================*/

__global__ void cuda_kernel_visco_PSM_2d_reconstruction_k_space
(
	float beta1, float beta2,
	int it, int nt, int ntx, int ntz, int nx, int nz, int L, float dx, float dz, float dt, 
	float *vp, float *Gamma, float averGamma, float f0, float Omega0, float alphaorder,
	float *kx, float *kz, 
	hipfftComplex *uk, hipfftComplex *uk0, 
	hipfftComplex *Lap_uk, hipfftComplex *amp_uk, hipfftComplex *pha_uk, hipfftComplex *sta_uk
)
{
	int bx=blockIdx.x;
	int by=blockIdx.y;
	int tx=threadIdx.x;
	int ty=threadIdx.y;
	int iz=by*BLOCK_SIZE+ty;
	int ix=bx*BLOCK_SIZE+tx;
	int ip=iz*ntx+ix;

	if(iz>=0&&iz<=ntz-1&&ix>=0&&ix<=ntx-1)
	{
		Lap_uk[ip].x=-(powf(kx[ix],2)+powf(kz[iz],2))*uk[ip].x;
		Lap_uk[ip].y=-(powf(kx[ix],2)+powf(kz[iz],2))*uk[ip].y;
		if(beta1!=0)
		{
			pha_uk[ip].x=powf((powf(kx[ix],2)+powf(kz[iz],2)), averGamma+1)*uk[ip].x;
			pha_uk[ip].y=powf((powf(kx[ix],2)+powf(kz[iz],2)), averGamma+1)*uk[ip].y;
		}
		if(beta2!=0)
		{
			amp_uk[ip].x=powf((powf(kx[ix],2)+powf(kz[iz],2)), averGamma+0.5)*(uk[ip].x-uk0[ip].x)/dt;
			amp_uk[ip].y=powf((powf(kx[ix],2)+powf(kz[iz],2)), averGamma+0.5)*(uk[ip].y-uk0[ip].y)/dt;
		}	
		if(beta2<0)
		{
			sta_uk[ip].x=powf((powf(kx[ix],2)+powf(kz[iz],2)), 0.5*alphaorder)*(uk[ip].x-uk0[ip].x)/dt;
			sta_uk[ip].y=powf((powf(kx[ix],2)+powf(kz[iz],2)), 0.5*alphaorder)*(uk[ip].y-uk0[ip].y)/dt;
		}

		uk0[ip].x=uk[ip].x;
		uk0[ip].y=uk[ip].y;			
	}
	__syncthreads();
}


//==========================================================
//  This subroutine is used for calculating reconstructed wavefileds in x-space
//  =========================================================
__global__ void cuda_kernel_visco_PSM_2d_reconstruction_x_space
(
	float beta1, float beta2,
	int it, int nt, int ntx, int ntz, int nx, int nz, int L, float dx, float dz, float dt, 
	float *vp, float *Gamma, float averGamma, float f0, float Omega0, float sigmafactor,
	float *ricker, int s_ix, int s_iz,
	hipfftComplex *u0, hipfftComplex *u1, hipfftComplex *u2,
	hipfftComplex *Lap, hipfftComplex *amp_Lap, hipfftComplex *pha_Lap, hipfftComplex *sta_Lap,
	float *borders_up, float *borders_bottom, float *borders_left, float *borders_right
)
{
	int bx=blockIdx.x;
	int by=blockIdx.y;
	int tx=threadIdx.x;
	int ty=threadIdx.y;
	int iz=by*BLOCK_SIZE+ty;
	int ix=bx*BLOCK_SIZE+tx;
	int ip=iz*ntx+ix;

	int icp;
	float eta, tau;

	if(iz>=0&&iz<=ntz-1&&ix>=0&&ix<=ntx-1)
	{		
		eta= -powf(vp[ip],2*Gamma[ip])*powf(Omega0,-2*Gamma[ip])*cos(Gamma[ip]*PI);
		tau= -powf(vp[ip],2*Gamma[ip]-1)*powf(Omega0,-2*Gamma[ip])*sin(Gamma[ip]*PI);

		// scale fft by dividing (ntx*ntz)
		Lap[ip].x=Lap[ip].x/(ntx*ntz);
		pha_Lap[ip].x=pha_Lap[ip].x/(ntx*ntz);
		amp_Lap[ip].x=amp_Lap[ip].x/(ntx*ntz);
		sta_Lap[ip].x=sta_Lap[ip].x/(ntx*ntz);

		u2[ip].x=powf(vp[ip]*cos(Gamma[ip]*PI/2),2)*powf(dt,2)
			*(
				Lap[ip].x
				+beta1*(eta*pha_Lap[ip].x-Lap[ip].x)
				+beta2*tau*amp_Lap[ip].x
				+tau*sigmafactor*sta_Lap[ip].x
			)
			+2*u1[ip].x-u0[ip].x;
	}

	// add borders 
	if(ix>=L&&ix<=ntx-L-1&&iz==L)
	{
		u2[ip].x=borders_up[it*nx+ix-L];
	}
	if(ix>=L&&ix<=ntx-L-1&&iz==ntz-L-1)
	{
		u2[ip].x=borders_bottom[it*nx+ix-L];
	}
	if(iz>=L&&iz<=ntz-L-1&&ix==L)
	{
		u2[ip].x=borders_left[it*nz+iz-L];
	}
	if(iz>=L&&iz<=ntz-L-1&&ix==ntx-L-1)
	{
		u2[ip].x=borders_right[it*nz+iz-L];
	}
	__syncthreads();
}


//==========================================================
//  This subroutine is used for reading checkpoints
//  =========================================================
__global__ void cuda_kernel_checkpoints_In
(
	int it, int nt, int ntx, int ntz, int nx, int nz, int L, float dx, float dz, float dt, 
	hipfftComplex *u1, hipfftComplex *u2,
	float *u_cp, int N_cp, int *t_cp
)
{
	int bx=blockIdx.x;
	int by=blockIdx.y;
	int tx=threadIdx.x;
	int ty=threadIdx.y;
	int iz=by*BLOCK_SIZE+ty;
	int ix=bx*BLOCK_SIZE+tx;
	int ip=iz*ntx+ix;

	int icp;

	for(icp=0;icp<N_cp;icp++)
	{
		if(icp%2==0&&it==t_cp[icp])
		{
			if(iz>=0&&iz<=ntz-1&&ix>=0&&ix<=ntx-1)
			{
				u2[ip].x=u_cp[icp*ntx*ntz+ip];
				u1[ip].x=u_cp[(icp+1)*ntx*ntz+ip];
			}
		}
	}
	__syncthreads();
}


//==========================================================
//  This subroutine is used for calculating backward wavefileds in k-space
//  =========================================================
__global__ void cuda_kernel_visco_PSM_2d_backward_k_space
(
	float beta1, float beta2,
	int it, int nt, int ntx, int ntz, float dx, float dz, float dt, 
	float *vp, float *Gamma, float averGamma, float f0, float Omega0, float alphaorder,
	float *kx, float *kz, 
	hipfftComplex *uk, hipfftComplex *uk0, 
	hipfftComplex *Lap_uk, hipfftComplex *amp_uk, hipfftComplex *pha_uk, hipfftComplex *sta_uk
)
{
	int bx=blockIdx.x;
	int by=blockIdx.y;
	int tx=threadIdx.x;
	int ty=threadIdx.y;
	int iz=by*BLOCK_SIZE+ty;
	int ix=bx*BLOCK_SIZE+tx;
	int ip=iz*ntx+ix;

	if(iz>=0&&iz<=ntz-1&&ix>=0&&ix<=ntx-1)
	{
		Lap_uk[ip].x=-(powf(kx[ix],2)+powf(kz[iz],2))*uk[ip].x;
		Lap_uk[ip].y=-(powf(kx[ix],2)+powf(kz[iz],2))*uk[ip].y;
		if(beta1!=0)
		{
			pha_uk[ip].x=powf((powf(kx[ix],2)+powf(kz[iz],2)), averGamma+1)*uk[ip].x;
			pha_uk[ip].y=powf((powf(kx[ix],2)+powf(kz[iz],2)), averGamma+1)*uk[ip].y;
		}
		if(beta2!=0)
		{
			amp_uk[ip].x=powf((powf(kx[ix],2)+powf(kz[iz],2)), averGamma+0.5)*(uk[ip].x-uk0[ip].x)/dt;
			amp_uk[ip].y=powf((powf(kx[ix],2)+powf(kz[iz],2)), averGamma+0.5)*(uk[ip].y-uk0[ip].y)/dt;
		}	
		if(beta2<0)
		{
			sta_uk[ip].x=powf((powf(kx[ix],2)+powf(kz[iz],2)), 0.5*alphaorder)*(uk[ip].x-uk0[ip].x)/dt;
			sta_uk[ip].y=powf((powf(kx[ix],2)+powf(kz[iz],2)), 0.5*alphaorder)*(uk[ip].y-uk0[ip].y)/dt;
		}

		uk0[ip].x=uk[ip].x;
		uk0[ip].y=uk[ip].y;			
	}
	__syncthreads();
}


//==========================================================
//  This subroutine is used for calculating backward wavefileds in x-space
//  ========================================================
__global__ void cuda_kernel_visco_PSM_2d_backward_x_space
(
	float beta1, float beta2,
	int it, int nt, int ntx, int ntz, float dx, float dz, float dt, 
	float *vp, float *Gamma, float averGamma, float f0, float Omega0, float sigmafactor,
	float *seismogram_rms, int *r_ix, int *r_iz, int s_ix, int s_iz, int rnmax, int nrx_obs,
	hipfftComplex *u0, hipfftComplex *u1, hipfftComplex *u2,
	hipfftComplex *Lap, hipfftComplex *amp_Lap, hipfftComplex *pha_Lap, hipfftComplex *sta_Lap
)
{
	int bx=blockIdx.x;
	int by=blockIdx.y;
	int tx=threadIdx.x;
	int ty=threadIdx.y;
	int iz=by*BLOCK_SIZE+ty;
	int ix=bx*BLOCK_SIZE+tx;
	int ip=iz*ntx+ix;

	float eta, tau;

	if(iz>=0&&iz<=ntz-1&&ix>=0&&ix<=ntx-1)
	{
		eta= -powf(vp[ip],2*Gamma[ip])*powf(Omega0,-2*Gamma[ip])*cos(Gamma[ip]*PI);
		tau= -powf(vp[ip],2*Gamma[ip]-1)*powf(Omega0,-2*Gamma[ip])*sin(Gamma[ip]*PI);

		// scaling fft 
		Lap[ip].x=Lap[ip].x/(ntx*ntz);
		pha_Lap[ip].x=pha_Lap[ip].x/(ntx*ntz);
		amp_Lap[ip].x=amp_Lap[ip].x/(ntx*ntz);
		sta_Lap[ip].x=sta_Lap[ip].x/(ntx*ntz);

		u2[ip].x=powf(vp[ip]*cos(Gamma[ip]*PI/2),2)*powf(dt,2)
			*(
				Lap[ip].x
				+beta1*(eta*pha_Lap[ip].x-Lap[ip].x)
				+beta2*tau*amp_Lap[ip].x
				+tau*sigmafactor*sta_Lap[ip].x
			)
			+2*u1[ip].x-u0[ip].x;	
	}

	// add seismogram as source
	int irx_min = s_ix-nrx_obs;
	int irx_max = s_ix+nrx_obs;
	if(irx_min<r_ix[0])
		irx_min = r_ix[0];
	if(irx_max>r_ix[rnmax-1])
		irx_max = r_ix[rnmax-1];

	if(ix>=0&&ix<=rnmax-1)
	{
		if(abs(s_iz-r_iz[ix])>=20)
			u2[r_iz[ix]*ntx + r_ix[ix]].x = seismogram_rms[it*rnmax+ix];
	}
	__syncthreads();
}


//==========================================================
//  This subroutine is used for imaging
// ========================================================
__global__ void cuda_kernel_image
(
	int ntx, int ntz, int L,
	hipfftComplex *u2_inv, hipfftComplex *u2,
	float *image_cor, float *image_sources, float *image_receivers
)
{
	int bx=blockIdx.x;
	int by=blockIdx.y;
	int tx=threadIdx.x;
	int ty=threadIdx.y;
	int iz=by*BLOCK_SIZE+ty;
	int ix=bx*BLOCK_SIZE+tx;
	int ip=iz*ntx+ix;

	if(iz>=L&&iz<=ntz-L-1&&ix>=L&&ix<=ntx-L-1)
	{
		image_cor[ip]+=u2_inv[ip].x*u2[ip].x;
		image_sources[ip]+=u2_inv[ip].x*u2_inv[ip].x;
		image_receivers[ip]+=u2[ip].x*u2[ip].x;    
	}
	__syncthreads();
}

//==========================================================
//  This subroutine is used for absorbing boundary condition
//  ========================================================
__global__ void cuda_kernel_MTF_2nd
(
	int L, int ntx, int ntz, float dx, float dz, float dt, 
	float *vp, hipfftComplex *u0, hipfftComplex *u1, hipfftComplex *u2
)
{
	int bx=blockIdx.x;
	int by=blockIdx.y;
	int tx=threadIdx.x;
	int ty=threadIdx.y;
	int iz=by*BLOCK_SIZE+ty;
	int ix=bx*BLOCK_SIZE+tx;
	int ip=iz*ntx+ix;

	int ipp=iz*ntx+(ntx-1-ix);
	int ippp=(ntz-1-iz)*ntx+ix;

	float alpha=1.0;
	float w, s, t1, t2, t3;

	// left ABC ...
	if(ix>=0&&ix<=L-1&&iz>=0&&iz<=ntz-1)
	{
		w=1-1.0*ix/L;
		s=alpha*vp[ip]*dt/dx;
		t1=(2-s)*(1-s)/2;
		t2=s*(2-s);
		t3=s*(s-1)/2;	

		u2[ip].x=w*
			(
				(1*2)*
				(
					t1*u1[ip].x+t2*u1[ip+1].x+t3*u1[ip+2].x
				)
				+(-1*1)*
				(
					t1*t1*u0[ip].x
					+2*t1*t2*u0[ip+1].x
					+(2*t1*t3+t2*t2)*u0[ip+2].x
					+2*t2*t3*u0[ip+3].x
					+t3*t3*u0[ip+4].x
				)
			)
			+(1-w)*u2[ip].x;								
	}

	// right ABC ...
	if(ix>=ntx-L&&ix<=ntx-1&&iz>=0&&iz<=ntz-1)
	{
		w=1-1.0*(ntx-1-ix)/L;

		s=alpha*vp[ip]*dt/dx;
		t1=(2-s)*(1-s)/2;
		t2=s*(2-s);
		t3=s*(s-1)/2;			

		u2[ip].x=w*
				(
					(1*2)*
					(
						t1*u1[ip].x
						+t2*u1[ip-1].x
						+t3*u1[ip-2].x
					)
					+(-1*1)*
					(
						t1*t1*u0[ip].x
						+2*t1*t2*u0[ip-1].x
						+(2*t1*t3+t2*t2)*u0[ip-2].x
						+2*t2*t3*u0[ip-3].x
						+t3*t3*u0[ip-4].x
					)
				)
				+(1-w)*u2[ip].x;								
	}


	// up ABC ...
	if(iz>=0&&iz<=L-1&&ix>=0&&ix<=ntx-1)
	{
		w=1-1.0*iz/L;
		s=alpha*vp[ip]*dt/dz;	
		t1=(2-s)*(1-s)/2;
		t2=s*(2-s);
		t3=s*(s-1)/2;

		u2[ip].x=w*
			(
				(1*2)*
				(
					t1*u1[ip].x
					+t2*u1[ip+ntx].x
					+t3*u1[ip+2*ntx].x
				)
				+(-1*1)*
				(
					t1*t1*u0[ip].x
					+2*t1*t2*u0[ip+ntx].x
					+(2*t1*t3+t2*t2)*u0[ip+2*ntx].x
					+2*t2*t3*u0[ip+3*ntx].x
					+t3*t3*u0[ip+4*ntx].x
				)
			)
			+(1-w)*u2[ip].x;			
	}

	// bottom ABC ...
	if(iz>=ntz-L&&iz<=ntz-1&&ix>=0&&ix<=ntx-1)
	{
		w=1-1.0*(ntz-1-iz)/L;
		s=alpha*vp[ip]*dt/dz;	
		t1=(2-s)*(1-s)/2;
		t2=s*(2-s);
		t3=s*(s-1)/2;

		u2[ip].x=w*
			(
				(1*2)*
				(
					t1*u1[ip].x
					+t2*u1[ip-ntx].x
					+t3*u1[ip-2*ntx].x
				)
				+(-1*1)*
				(
					t1*t1*u0[ip].x
					+2*t1*t2*u0[ip-ntx].x
					+(2*t1*t3+t2*t2)*u0[ip-2*ntx].x
					+2*t2*t3*u0[ip-3*ntx].x
					+t3*t3*u0[ip-4*ntx].x
				)
			)
			+(1-w)*u2[ip].x;		
	}
	__syncthreads();	
}







//==========================================================
//  This subroutine are used for forward modeling
//	For more details please refer to Eq(1) in our paper
//  =========================================================
extern "C"
void cuda_visco_PSM_2d_forward
(
	int beta1, int beta2,
	int nt, int ntx, int ntz, int ntp, int nx, int nz, int L, float dx, float dz, float dt,
	float *vp, float *Gamma, float avervp, float averGamma, float f0, float Omega0, float *ricker,
	int myid, int is, struct Source ss[], struct MultiGPU plan[], int GPU_N, int rnmax, int nrx_obs, int N_cp, int *t_cp,
	float alphaorder, float sigmafactor,
	int Sto_Rec, int vp_type, int Save_Not
)
{
	int i, it, ix, iz, ip, icp;
	size_t size_model=sizeof(float)*ntp;
	FILE *fp;
	char filename[40];
	float *u2_real;
	u2_real = (float*)malloc(sizeof(float)*ntp);

	// define multistream  variable
	Multistream plans[GPU_N];

	// define streaming cufft handle (very important!!!)
	for(i=0;i<GPU_N;i++)
	{
		hipSetDevice(i);
		hipStreamCreate(&plans[i].stream);	
		hipfftSetStream(plan[i].PLAN_FORWARD,plans[i].stream);
		hipfftSetStream(plan[i].PLAN_BACKWARD,plans[i].stream);
	}	

	// block size 16*16; 
	dim3 dimBlock(BLOCK_SIZE,BLOCK_SIZE);
	// grid size ntx/16*ntz/16
	dim3 dimGrid((ntx+dimBlock.x-1)/dimBlock.x,(ntz+dimBlock.y-1)/dimBlock.y);

	// copy the vectors from the host to the device
	for(i=0;i<GPU_N;i++)
	{
		hipSetDevice(i);
		hipMemcpyAsync(plan[i].d_r_ix,ss[is+i].r_ix,sizeof(int)*rnmax,hipMemcpyHostToDevice,plans[i].stream);
		hipMemcpyAsync(plan[i].d_r_iz,ss[is+i].r_iz,sizeof(int)*rnmax,hipMemcpyHostToDevice,plans[i].stream);
		hipMemcpyAsync(plan[i].d_ricker,ricker,sizeof(float)*nt,hipMemcpyHostToDevice,plans[i].stream);
		hipMemcpyAsync(plan[i].d_vp,vp,size_model,hipMemcpyHostToDevice,plans[i].stream);
		hipMemcpyAsync(plan[i].d_Gamma,Gamma,size_model,hipMemcpyHostToDevice,plans[i].stream);
		hipMemcpyAsync(plan[i].d_t_cp,t_cp,N_cp*sizeof(int),hipMemcpyHostToDevice,plans[i].stream);
	}

	// initializing wavefield variables and define k variables
	for(i=0;i<GPU_N;i++)
	{
		hipSetDevice(i);
		cuda_kernel_initialization<<<dimGrid,dimBlock,0,plans[i].stream>>>
			(ntx, ntz, plan[i].d_u0, plan[i].d_u1, plan[i].d_u2, plan[i].d_uk0, plan[i].d_uk, 
			plan[i].d_Lap, plan[i].d_amp_Lap, plan[i].d_pha_Lap, plan[i].d_sta_Lap);
		cuda_kernel_k_define<<<dimGrid,dimBlock,0,plans[i].stream>>>
			(ntx, ntz, dx, dz, plan[i].d_kx, plan[i].d_kz);
	}

	// forward time iteration
	for(it=0;it<nt;it++)  
	{
		for(i=0;i<GPU_N;i++)
		{
			hipSetDevice(i);
			hipfftExecC2C(plan[i].PLAN_FORWARD,plan[i].d_u1,plan[i].d_uk,HIPFFT_FORWARD); //HIPFFT_FORWARD

			cuda_kernel_visco_PSM_2d_forward_k_space<<<dimGrid,dimBlock,0,plans[i].stream>>>
				(
					beta1, beta2,
					it, nt, ntx, ntz, dx, dz, dt, 
					plan[i].d_vp, plan[i].d_Gamma, averGamma, f0, Omega0, alphaorder,
					plan[i].d_kx, plan[i].d_kz,  
					plan[i].d_uk, plan[i].d_uk0, 
					plan[i].d_Lap_uk, plan[i].d_amp_uk, plan[i].d_pha_uk, plan[i].d_sta_uk
				);		

			hipfftExecC2C(plan[i].PLAN_BACKWARD,plan[i].d_Lap_uk,plan[i].d_Lap,HIPFFT_BACKWARD); //HIPFFT_BACKWARD

			if(beta1!=0)
			{	
				hipfftExecC2C(plan[i].PLAN_BACKWARD,plan[i].d_pha_uk,plan[i].d_pha_Lap,HIPFFT_BACKWARD); //HIPFFT_BACKWARD					
			}

			if(beta2!=0)
			{						
				hipfftExecC2C(plan[i].PLAN_BACKWARD,plan[i].d_amp_uk,plan[i].d_amp_Lap,HIPFFT_BACKWARD); //HIPFFT_BACKWARD
				hipfftExecC2C(plan[i].PLAN_BACKWARD,plan[i].d_sta_uk,plan[i].d_sta_Lap,HIPFFT_BACKWARD); //HIPFFT_BACKWARD
			}

			cuda_kernel_visco_PSM_2d_forward_x_space<<<dimGrid,dimBlock,0,plans[i].stream>>>
				(
					beta1, beta2,
					it, nt, ntx, ntz, nx, nz, L, dx, dz, dt, 
					plan[i].d_vp, plan[i].d_Gamma, averGamma, f0, Omega0, sigmafactor,
					plan[i].d_seismogram, plan[i].d_r_ix, plan[i].d_r_iz, rnmax, plan[i].d_ricker, ss[is+i].s_ix, ss[is+i].s_iz,
					plan[i].d_u0, plan[i].d_u1, plan[i].d_u2,
					plan[i].d_Lap, plan[i].d_amp_Lap, plan[i].d_pha_Lap, plan[i].d_sta_Lap,
					plan[i].d_borders_up, plan[i].d_borders_bottom, plan[i].d_borders_left, plan[i].d_borders_right,
					plan[i].d_u2_final0, plan[i].d_u2_final1,
					Sto_Rec, vp_type				
				);
		
		
			// MTF absorbing boundary condition
			cuda_kernel_MTF_2nd<<<dimGrid,dimBlock,0,plans[i].stream>>>
				(L, ntx, ntz, dx, dz, dt, plan[i].d_vp, plan[i].d_u0, plan[i].d_u1, plan[i].d_u2);

			// record wavefields at checkpoints
			if(Sto_Rec==0&&vp_type==2)
			{
				cuda_kernel_checkpoints_Out<<<dimGrid,dimBlock,0,plans[i].stream>>>
					(
						it, nt, ntx, ntz, nx, nz, L, dx, dz, dt, 
						plan[i].d_u1, plan[i].d_u2,
						plan[i].d_u_cp, N_cp, plan[i].d_t_cp			
					);
			}

			// write wavefields at checkpoints and last two time steps
			if(Sto_Rec==1&&vp_type==2||Save_Not==1)
			{
				hipMemcpyAsync(plan[i].u2,plan[i].d_u2,sizeof(hipfftComplex)*ntp,hipMemcpyDeviceToHost,plans[i].stream);

				sprintf(filename,"./output/GPU_%d_u2_%d.dat",i,it);     
				fp=fopen(filename,"wb");
				for(ix=0;ix<ntx-0;ix++)
				{
					for(iz=0;iz<ntz-0;iz++)
					{
						u2_real[iz*ntx+ix]=plan[i].u2[iz*ntx+ix].x;			
						fwrite(&u2_real[iz*ntx+ix],sizeof(float),1,fp);
					}
				}
				fclose(fp);			
			}

			// updating wavefields
			cuda_kernel_update<<<dimGrid,dimBlock,0,plans[i].stream>>>
				(ntx, ntz, plan[i].d_u0, plan[i].d_u1, plan[i].d_u2);

			if(myid==0&&it%100==0)
			{
				printf("shot %d forward %d has finished!\n", is+i+1, it);
			}

		}// GPU_N end	
	}// nt end



	for(i=0;i<GPU_N;i++)
	{
		hipSetDevice(i);

		// copy seismograms to host memory
		if(vp_type==0)	// homogeneous model
		{
			hipMemcpyAsync(plan[i].seismogram_dir,plan[i].d_seismogram,
					sizeof(float)*ss[is+i].r_n*nt,hipMemcpyDeviceToHost,plans[i].stream);
		}
		else if(vp_type==1)	// ture model
		{
			hipMemcpyAsync(plan[i].seismogram_obs,plan[i].d_seismogram,
					sizeof(float)*ss[is+i].r_n*nt,hipMemcpyDeviceToHost,plans[i].stream);
		}
		else if(vp_type==2)	// initial model
		{
			hipMemcpyAsync(plan[i].seismogram_syn,plan[i].d_seismogram,
					sizeof(float)*ss[is+i].r_n*nt,hipMemcpyDeviceToHost,plans[i].stream);
		}
	}

	for(i=0;i<GPU_N;i++)
	{
		hipSetDevice(i);
		hipDeviceSynchronize();
	}

	//free the memory of DEVICE
	for(i=0;i<GPU_N;i++)
	{
		hipSetDevice(i);	
		hipStreamDestroy(plans[i].stream);
	}
	free(u2_real);
}



//==========================================================
//  This subroutine are used for backward modeling
//	For more details please refer to Eq(5) in our paper
//  =========================================================
extern "C"
void cuda_visco_PSM_2d_backward
(
	int beta1, int beta2,
	int nt, int ntx, int ntz, int ntp, int nx, int nz, int L, float dx, float dz, float dt,
	float *vp, float *Gamma, float avervp, float averGamma, float f0, float Omega0, float *ricker,
	int myid, int is, struct Source ss[], struct MultiGPU plan[], int GPU_N, int rnmax, int nrx_obs, int N_cp, int *t_cp,
	float alphaorder, float sigmafactor,
	int Sto_Rec, int Save_Not
)
{
	int i, it, ix, iz, ip;
	size_t size_model=sizeof(float)*ntp;
	FILE *fp;
	char filename[40];
	float *u2_real;
	u2_real = (float*)malloc(sizeof(float)*ntp);

	// define multistream  variable
	Multistream plans[GPU_N];

	// define streaming cufft handle (very important!!!)
	for(i=0;i<GPU_N;i++)
	{
		hipSetDevice(i);
		hipStreamCreate(&plans[i].stream);	
		hipfftSetStream(plan[i].PLAN_FORWARD,plans[i].stream);
		hipfftSetStream(plan[i].PLAN_BACKWARD,plans[i].stream);
	}	

	dim3 dimBlock(BLOCK_SIZE,BLOCK_SIZE);
	dim3 dimGrid((ntx+dimBlock.x-1)/dimBlock.x,(ntz+dimBlock.y-1)/dimBlock.y);


	for(i=0;i<GPU_N;i++)
	{
		hipSetDevice(i);

		// initializating wavefields for reconstruction
		cuda_kernel_initialization<<<dimGrid,dimBlock,0,plans[i].stream>>>
			(ntx, ntz, plan[i].d_u0_inv, plan[i].d_u1_inv, plan[i].d_u2_inv, plan[i].d_uk0_inv, plan[i].d_uk_inv,
			plan[i].d_Lap, plan[i].d_amp_Lap, plan[i].d_pha_Lap, plan[i].d_sta_Lap);

		// initializating last two wavefields for backward propagation
		cuda_kernel_initialization_Finals<<<dimGrid,dimBlock,0,plans[i].stream>>>
			(ntx, ntz, plan[i].d_u0_inv, plan[i].d_u1_inv, plan[i].d_u2_final0, plan[i].d_u2_final1);

		// initialization wavefield for backward propagation
		cuda_kernel_initialization<<<dimGrid,dimBlock,0,plans[i].stream>>>
			(ntx, ntz, plan[i].d_u0, plan[i].d_u1, plan[i].d_u2, plan[i].d_uk0, plan[i].d_uk,
			plan[i].d_Lap, plan[i].d_amp_Lap, plan[i].d_pha_Lap, plan[i].d_sta_Lap);

		// initialization image variables for imaging
		cuda_kernel_initialization_images<<<dimGrid,dimBlock,0,plans[i].stream>>>
			(ntx, ntz, plan[i].d_image_cor, plan[i].d_image_nor, plan[i].d_image_sources, plan[i].d_image_receivers);			
	}

	// copy the vectors from the host to the device
	for(i=0;i<GPU_N;i++)
	{
		hipSetDevice(i);
		hipMemcpyAsync(plan[i].d_seismogram_rms,plan[i].seismogram_rms,
			sizeof(float)*rnmax*nt,hipMemcpyHostToDevice,plans[i].stream);
	}

	int beta1_inv=beta1;
	int beta2_inv=-1*beta2;		// source wavefield is compensated, so reconstruction process is attenuated



	// backward time iteration
	for(it=nt-3;it>=0;it--)  
	{
		for(i=0;i<GPU_N;i++)
		{
			hipSetDevice(i);

			// reconstructing source wavefield using checkpointing scheme
			if(Sto_Rec==0)
			{
				hipfftExecC2C(plan[i].PLAN_FORWARD,plan[i].d_u1_inv,plan[i].d_uk_inv,HIPFFT_FORWARD); //HIPFFT_FORWARD
				cuda_kernel_visco_PSM_2d_reconstruction_k_space<<<dimGrid,dimBlock,0,plans[i].stream>>>
					(
						beta1_inv, beta2_inv,
						it, nt, ntx, ntz, nx, nz, L, dx, dz, dt, 
						plan[i].d_vp, plan[i].d_Gamma, averGamma, f0, Omega0, alphaorder,
						plan[i].d_kx, plan[i].d_kz,  
						plan[i].d_uk_inv, plan[i].d_uk0_inv, 
						plan[i].d_Lap_uk, plan[i].d_amp_uk, plan[i].d_pha_uk, plan[i].d_sta_uk
					);	

				hipfftExecC2C(plan[i].PLAN_BACKWARD,plan[i].d_Lap_uk,plan[i].d_Lap,HIPFFT_BACKWARD); //HIPFFT_BACKWARD

				if(beta1_inv!=0)
				{	
					hipfftExecC2C(plan[i].PLAN_BACKWARD,plan[i].d_pha_uk,plan[i].d_pha_Lap,HIPFFT_BACKWARD); //HIPFFT_BACKWARD					
				}
		
				if(beta2_inv!=0)
				{
					hipfftExecC2C(plan[i].PLAN_BACKWARD,plan[i].d_amp_uk,plan[i].d_amp_Lap,HIPFFT_BACKWARD); //HIPFFT_BACKWARD
					hipfftExecC2C(plan[i].PLAN_BACKWARD,plan[i].d_sta_uk,plan[i].d_sta_Lap,HIPFFT_BACKWARD); //HIPFFT_BACKWARD
				}
				
				cuda_kernel_visco_PSM_2d_reconstruction_x_space<<<dimGrid,dimBlock,0,plans[i].stream>>>
					(
						beta1_inv, beta2_inv,
						it, nt, ntx, ntz, nx, nz, L, dx, dz, dt, 
						plan[i].d_vp, plan[i].d_Gamma, averGamma, f0, Omega0, sigmafactor,
						plan[i].d_ricker, ss[is+i].s_ix, ss[is+i].s_iz,
						plan[i].d_u0_inv, plan[i].d_u1_inv, plan[i].d_u2_inv,
						plan[i].d_Lap, plan[i].d_amp_Lap, plan[i].d_pha_Lap, plan[i].d_sta_Lap,			
						plan[i].d_borders_up, plan[i].d_borders_bottom, plan[i].d_borders_left, plan[i].d_borders_right
					);	


				// MTF absorbing boundary condition
				cuda_kernel_MTF_2nd<<<dimGrid,dimBlock,0,plans[i].stream>>>
					(L, ntx, ntz, dx, dz, dt, plan[i].d_vp, plan[i].d_u0_inv, plan[i].d_u1_inv, plan[i].d_u2_inv);

				// read wavefields at checkpoints
				cuda_kernel_checkpoints_In<<<dimGrid,dimBlock,0,plans[i].stream>>>
					(
						it, nt, ntx, ntz, nx, nz, L, dx, dz, dt, 
						plan[i].d_u1_inv, plan[i].d_u2_inv,
						plan[i].d_u_cp, N_cp, plan[i].d_t_cp			
					);							

				// updating wavefields
				cuda_kernel_update<<<dimGrid,dimBlock,0,plans[i].stream>>>
					(ntx, ntz, plan[i].d_u0_inv, plan[i].d_u1_inv, plan[i].d_u2_inv);
			}

			// read source wavefields from disk 
			if(Sto_Rec==1)
			{
				sprintf(filename,"./output/GPU_%d_u2_%d.dat",i,it); 
				fp=fopen(filename,"rb");
				for(ix=0;ix<ntx-0;ix++)
				{
					for(iz=0;iz<ntz-0;iz++)
					{								
						fread(&u2_real[iz*ntx+ix],sizeof(float),1,fp);
						plan[i].u2[iz*ntx+ix].x=u2_real[iz*ntx+ix];	
						plan[i].u2[iz*ntx+ix].y=0.0;
					}
				}
				fclose(fp);
				hipMemcpyAsync(plan[i].d_u2_inv,plan[i].u2,sizeof(hipfftComplex)*ntp,hipMemcpyHostToDevice,plans[i].stream);
			}


			// backward propagation for imaging
			hipfftExecC2C(plan[i].PLAN_FORWARD,plan[i].d_u1,plan[i].d_uk,HIPFFT_FORWARD); //HIPFFT_FORWARD

			cuda_kernel_visco_PSM_2d_backward_k_space<<<dimGrid,dimBlock,0,plans[i].stream>>>
				(
					beta1, beta2,
					it, nt, ntx, ntz, dx, dz, dt, 
					plan[i].d_vp, plan[i].d_Gamma, averGamma, f0, Omega0, alphaorder,
					plan[i].d_kx, plan[i].d_kz,  
					plan[i].d_uk, plan[i].d_uk0, 
					plan[i].d_Lap_uk, plan[i].d_amp_uk, plan[i].d_pha_uk, plan[i].d_sta_uk
				);

			hipfftExecC2C(plan[i].PLAN_BACKWARD,plan[i].d_Lap_uk,plan[i].d_Lap,HIPFFT_BACKWARD); //HIPFFT_BACKWARD

			if(beta1!=0)
			{	
				hipfftExecC2C(plan[i].PLAN_BACKWARD,plan[i].d_pha_uk,plan[i].d_pha_Lap,HIPFFT_BACKWARD); //HIPFFT_BACKWARD					
			}

			if(beta2!=0)
			{
				hipfftExecC2C(plan[i].PLAN_BACKWARD,plan[i].d_amp_uk,plan[i].d_amp_Lap,HIPFFT_BACKWARD); //HIPFFT_BACKWARD
				hipfftExecC2C(plan[i].PLAN_BACKWARD,plan[i].d_sta_uk,plan[i].d_sta_Lap,HIPFFT_BACKWARD); //HIPFFT_BACKWARD
			}

			cuda_kernel_visco_PSM_2d_backward_x_space<<<dimGrid,dimBlock,0,plans[i].stream>>>
				(
					beta1, beta2,
					it, nt, ntx, ntz, dx, dz, dt, 
					plan[i].d_vp, plan[i].d_Gamma, averGamma, f0, Omega0, sigmafactor,
					plan[i].d_seismogram_rms, plan[i].d_r_ix, plan[i].d_r_iz, ss[is+i].s_ix, ss[is+i].s_iz, rnmax, nrx_obs,
					plan[i].d_u0, plan[i].d_u1, plan[i].d_u2,
					plan[i].d_Lap, plan[i].d_amp_Lap, plan[i].d_pha_Lap, plan[i].d_sta_Lap
				);
				
			
			// MTF absorbing boundary condition
			cuda_kernel_MTF_2nd<<<dimGrid,dimBlock,0,plans[i].stream>>>
				(L, ntx, ntz, dx, dz, dt, plan[i].d_vp, plan[i].d_u0, plan[i].d_u1, plan[i].d_u2);
		
			// imaging (exclude duration of explosion)
			int it0 = int(2/(f0*dt));
			if (it>it0)
			{
				cuda_kernel_image<<<dimGrid,dimBlock,0,plans[i].stream>>>
					(
						ntx, ntz, L,
						plan[i].d_u2_inv, plan[i].d_u2,
						plan[i].d_image_cor, plan[i].d_image_sources, plan[i].d_image_receivers
					);			
			}


			// write backward wavefields and reconstructed wavefields to disk
			if(Save_Not==1)
			{
				hipMemcpyAsync(plan[i].u1,plan[i].d_u2_inv,sizeof(hipfftComplex)*ntp,hipMemcpyDeviceToHost,plans[i].stream);
				hipMemcpyAsync(plan[i].u2,plan[i].d_u2,sizeof(hipfftComplex)*ntp,hipMemcpyDeviceToHost,plans[i].stream);
				hipStreamSynchronize(plans[i].stream);
				sprintf(filename,"./output/GPU_%d_u2_inv_%d.dat",i,it); 
				fp=fopen(filename,"wb");
				for(ix=0;ix<ntx-0;ix++)
				{
					for(iz=0;iz<ntz-0;iz++)
					{
						u2_real[iz*ntx+ix]=plan[i].u1[iz*ntx+ix].x;
						fwrite(&u2_real[iz*ntx+ix],sizeof(float),1,fp);
					}
				}
				fclose(fp);
				sprintf(filename,"./output/GPU_%d_u2_bak_%d.dat",i,it); 
				fp=fopen(filename,"wb");
				for(ix=0;ix<ntx-0;ix++)
				{
					for(iz=0;iz<ntz-0;iz++)
					{	
						u2_real[iz*ntx+ix]=plan[i].u2[iz*ntx+ix].x;				
						fwrite(&u2_real[iz*ntx+ix],sizeof(float),1,fp);
					}
				}
				fclose(fp);
			}

			// updating wavefields 
			cuda_kernel_update<<<dimGrid,dimBlock,0,plans[i].stream>>>
				(ntx, ntz, plan[i].d_u0, plan[i].d_u1, plan[i].d_u2);

			if(myid==0&&it%100==0)
			{
				printf("shot %d reconstruction and backward %d has finished!\n", is+i+1, it);
			}
		}// GPU_N end	
	}// nt end

	for(i=0;i<GPU_N;i++)
	{
		hipSetDevice(i);
		// output images 
		hipMemcpyAsync(plan[i].image_cor,plan[i].d_image_cor,sizeof(float)*ntp,hipMemcpyDeviceToHost,plans[i].stream);
		hipMemcpyAsync(plan[i].image_sources,plan[i].d_image_sources,sizeof(float)*ntp,hipMemcpyDeviceToHost,plans[i].stream);
		hipMemcpyAsync(plan[i].image_receivers,plan[i].d_image_receivers,sizeof(float)*ntp,hipMemcpyDeviceToHost,plans[i].stream);
	}

	for(i=0;i<GPU_N;i++)
	{
		hipSetDevice(i);
		hipDeviceSynchronize();
	}

	//free the memory of DEVICE
	for(i=0;i<GPU_N;i++)
	{
		hipSetDevice(i);
		hipStreamDestroy(plans[i].stream);
	}
	free(u2_real);	
}


//==========================================================
//  This two subroutines are used for Laplace filteing
//  ========================================================
extern "C"
void Laplace_filtering
(
	float *image, int ntx, int ntz, float dx, float dz
)
{ 
	int ix,iz,ip,K,NX,NZ;

	K=(int)ceil(log(1.0*ntx)/log(2.0));
	NX=(int)pow(2.0,K);

	K=(int)ceil(log(1.0*ntz)/log(2.0));
	NZ=(int)pow(2.0,K);

	float dkx,dkz;
	float kx,kz;

	dkx=(float)1.0/((NX)*dx);
	dkz=(float)1.0/((NZ)*dz);

	int NTP=NX*NZ;

	hipfftComplex *pp,*temp,*tempout;		

	hipHostMalloc((void **)&pp, sizeof(hipfftComplex)*NX*NZ);
	hipMalloc((void **)&temp,sizeof(hipfftComplex)*NX*NZ);
	hipMalloc((void **)&tempout,sizeof(hipfftComplex)*NX*NZ);

	hipfftHandle plan;
	hipfftPlan2d(&plan,NX,NZ,HIPFFT_C2C);

	for(ip=0;ip<NTP;ip++)
	{ 
		pp[ip].x=0.0;
		pp[ip].y=0.0; 
	} 

	for(ix=0;ix<ntx;ix++)
	{            
		for(iz=0;iz<ntz;iz++)
		{
			pp[ix*NZ+iz].x=image[iz*ntx+ix];
		}
	} 

	hipMemcpy(temp,pp,sizeof(hipfftComplex)*NX*NZ,hipMemcpyHostToDevice);
	hipfftExecC2C(plan,temp,tempout,HIPFFT_FORWARD);
	hipMemcpy(pp,tempout,sizeof(hipfftComplex)*NX*NZ,hipMemcpyDeviceToHost);

	for(ix=0;ix<NX;ix++)
	{            
		for(iz=0;iz<NZ;iz++)
		{
			if(ix<NX/2)
			{
				kx=2*PI*ix*dkx;
			}
			if(ix>NX/2)	
			{
				kx=2*PI*(NX-1-ix)*dkx;
			}

			if(iz<NZ/2)
			{
				kz=2*PI*iz*dkz;//2*PI*(NZ/2-1-iz)*dkz;//0.0;//
			}
			if(iz>NZ/2)
			{
				kz=2*PI*(NZ-1-iz)*dkz;//2*PI*(iz-NZ/2)*dkz;//0.0;//
			}

			ip=ix*NZ+iz;

			pp[ip].x=pp[ip].x*(kx*kx+kz*kz);
			pp[ip].y=pp[ip].y*(kx*kx+kz*kz);

		}
	} 

	hipMemcpy(temp,pp,sizeof(hipfftComplex)*NX*NZ,hipMemcpyHostToDevice);
	hipfftExecC2C(plan,temp,tempout,HIPFFT_BACKWARD);
	hipMemcpy(pp,tempout,sizeof(hipfftComplex)*NX*NZ,hipMemcpyDeviceToHost);

	for(ix=0;ix<ntx;ix++)
	{            
		for(iz=0;iz<ntz;iz++)
		{
			image[iz*ntx+ix]=pp[ix*NZ+iz].x/(NX*NZ);
		}
	} 
	hipHostFree(pp);
	hipFree(temp);
	hipFree(tempout);
	hipfftDestroy(plan);

	return;
}


//=========================================================
//  Allocate the memory for variables in device
//  =======================================================
extern "C"
void cuda_Device_malloc
(
	int ntx, int ntz, int ntp, int nx, int nz, int nt, 
	float dx, float dz, int L, int rnmax, int N_cp,
	struct MultiGPU plan[], int GPU_N
)
{
	int i;
	size_t size_model=sizeof(float)*ntp;

	for(i=0;i<GPU_N;i++)
	{
		hipSetDevice(i);

		hipfftPlan2d(&plan[i].PLAN_FORWARD,ntz,ntx,HIPFFT_C2C);
		hipfftPlan2d(&plan[i].PLAN_BACKWARD,ntz,ntx,HIPFFT_C2C);

		hipHostMalloc((void **)&plan[i].u0, sizeof(hipfftComplex)*ntp);		
		hipHostMalloc((void **)&plan[i].u1, sizeof(hipfftComplex)*ntp);
		hipHostMalloc((void **)&plan[i].u2, sizeof(hipfftComplex)*ntp);		

		hipHostMalloc((void **)&plan[i].seismogram_obs, sizeof(float)*nt*rnmax);
		hipHostMalloc((void **)&plan[i].seismogram_dir, sizeof(float)*nt*rnmax);
		hipHostMalloc((void **)&plan[i].seismogram_syn, sizeof(float)*nt*rnmax);
		hipHostMalloc((void **)&plan[i].seismogram_rms, sizeof(float)*nt*rnmax);
		
		hipHostMalloc((void **)&plan[i].image_sources, sizeof(float)*ntp);
		hipHostMalloc((void **)&plan[i].image_receivers, sizeof(float)*ntp);
		hipHostMalloc((void **)&plan[i].image_cor, sizeof(float)*ntp);
		hipHostMalloc((void **)&plan[i].image_nor, sizeof(float)*ntp);

		hipMalloc((void**)&plan[i].d_r_ix,sizeof(int)*rnmax);
		hipMalloc((void**)&plan[i].d_r_iz,sizeof(int)*rnmax);
		hipMalloc((void**)&plan[i].d_ricker,sizeof(float)*nt);        //ricker

		hipMalloc((void**)&plan[i].d_vp,size_model);
		hipMalloc((void**)&plan[i].d_Gamma,size_model);

		hipMalloc((void**)&plan[i].d_u0,sizeof(hipfftComplex)*ntp);
		hipMalloc((void**)&plan[i].d_u1,sizeof(hipfftComplex)*ntp);
		hipMalloc((void**)&plan[i].d_u2,sizeof(hipfftComplex)*ntp);

		hipMalloc((void**)&plan[i].d_u0_inv,sizeof(hipfftComplex)*ntp);
		hipMalloc((void**)&plan[i].d_u1_inv,sizeof(hipfftComplex)*ntp);
		hipMalloc((void**)&plan[i].d_u2_inv,sizeof(hipfftComplex)*ntp);

		hipMalloc((void**)&plan[i].d_t_cp,sizeof(int)*N_cp);
		hipMalloc((void**)&plan[i].d_u_cp,size_model*N_cp);		//checkpoints

		hipMalloc((void**)&plan[i].d_kx,sizeof(float)*ntx);
		hipMalloc((void**)&plan[i].d_kz,sizeof(float)*ntz);

		hipMalloc((void **)&plan[i].d_uk,sizeof(hipfftComplex)*ntp);
		hipMalloc((void **)&plan[i].d_uk0,sizeof(hipfftComplex)*ntp);

		hipMalloc((void **)&plan[i].d_uk_inv,sizeof(hipfftComplex)*ntp);
		hipMalloc((void **)&plan[i].d_uk0_inv,sizeof(hipfftComplex)*ntp);

		hipMalloc((void **)&plan[i].d_Lap_uk,sizeof(hipfftComplex)*ntp);
		hipMalloc((void **)&plan[i].d_amp_uk,sizeof(hipfftComplex)*ntp);
		hipMalloc((void **)&plan[i].d_pha_uk,sizeof(hipfftComplex)*ntp);
		hipMalloc((void **)&plan[i].d_sta_uk,sizeof(hipfftComplex)*ntp);

		hipMalloc((void **)&plan[i].d_Lap,sizeof(hipfftComplex)*ntp);
		hipMalloc((void **)&plan[i].d_amp_Lap,sizeof(hipfftComplex)*ntp);
		hipMalloc((void **)&plan[i].d_pha_Lap,sizeof(hipfftComplex)*ntp);
		hipMalloc((void **)&plan[i].d_sta_Lap,sizeof(hipfftComplex)*ntp);
							
		hipMalloc((void**)&plan[i].d_seismogram,sizeof(float)*nt*rnmax);
		hipMalloc((void**)&plan[i].d_seismogram_rms,sizeof(float)*nt*rnmax);

		hipMalloc((void**)&plan[i].d_borders_up,sizeof(float)*nt*nx);
		hipMalloc((void**)&plan[i].d_borders_bottom,sizeof(float)*nt*nx);
		hipMalloc((void**)&plan[i].d_borders_left,sizeof(float)*nt*nz);
		hipMalloc((void**)&plan[i].d_borders_right,sizeof(float)*nt*nz);

		hipMalloc((void**)&plan[i].d_u2_final0,size_model);
		hipMalloc((void**)&plan[i].d_u2_final1,size_model);

		hipMalloc((void**)&plan[i].d_image_sources,size_model);
		hipMalloc((void**)&plan[i].d_image_receivers,size_model);
		hipMalloc((void**)&plan[i].d_image_cor,size_model);
		hipMalloc((void**)&plan[i].d_image_nor,size_model);
	}
}


//=========================================================
//  Free the memory for variables in device
//  =======================================================
extern "C"
void cuda_Device_free
(
	int ntx, int ntz, int ntp, int nx, int nz, int nt, 
	float dx, float dz, int L, int rnmax, int N_cp,
	struct MultiGPU plan[], int GPU_N
)
{
	int i;
	 

	for(i=0;i<GPU_N;i++)
	{
		hipSetDevice(i);

		hipfftDestroy(plan[i].PLAN_FORWARD);
		hipfftDestroy(plan[i].PLAN_BACKWARD);

		hipHostFree(plan[i].u0);
		hipHostFree(plan[i].u1);
		hipHostFree(plan[i].u2); 

		hipHostFree(plan[i].seismogram_obs);
		hipHostFree(plan[i].seismogram_dir);
		hipHostFree(plan[i].seismogram_syn); 
		hipHostFree(plan[i].seismogram_rms);
 
		hipHostFree(plan[i].image_cor);
		hipHostFree(plan[i].image_nor);
		hipHostFree(plan[i].image_sources);
		hipHostFree(plan[i].image_receivers);

		hipFree(plan[i].d_r_ix);
		hipFree(plan[i].d_r_iz);
		hipFree(plan[i].d_ricker);

		hipFree(plan[i].d_vp);
		hipFree(plan[i].d_Gamma);

		hipFree(plan[i].d_u0);
		hipFree(plan[i].d_u1);
		hipFree(plan[i].d_u2);

		hipFree(plan[i].d_u0_inv);
		hipFree(plan[i].d_u1_inv);
		hipFree(plan[i].d_u2_inv);

		hipFree(plan[i].d_t_cp);
		hipFree(plan[i].d_u_cp);

		hipFree(plan[i].d_kx);
		hipFree(plan[i].d_kz);

		hipFree(plan[i].d_uk);
		hipFree(plan[i].d_uk0);

		hipFree(plan[i].d_uk_inv);
		hipFree(plan[i].d_uk0_inv);

		hipFree(plan[i].d_Lap_uk);
		hipFree(plan[i].d_amp_uk);
		hipFree(plan[i].d_pha_uk);
		hipFree(plan[i].d_sta_uk);

		hipFree(plan[i].d_Lap);
		hipFree(plan[i].d_amp_Lap);
		hipFree(plan[i].d_pha_Lap);
		hipFree(plan[i].d_sta_Lap);

		hipFree(plan[i].d_seismogram);
		hipFree(plan[i].d_seismogram_rms);

		hipFree(plan[i].d_borders_up);
		hipFree(plan[i].d_borders_bottom);
		hipFree(plan[i].d_borders_left);
		hipFree(plan[i].d_borders_right);

		hipFree(plan[i].d_u2_final0);
		hipFree(plan[i].d_u2_final1);

		hipFree(plan[i].d_image_sources);
		hipFree(plan[i].d_image_receivers);
		hipFree(plan[i].d_image_cor);
		hipFree(plan[i].d_image_nor);
	}
}


//=========================================================
//  Initializating the memory for variables in device
//  =======================================================
extern "C"
void cuda_Host_initialization
(
	int ntx, int ntz, int ntp, int nx, int nz, int nt, 
	float dx, float dz, int L, int rnmax, int N_cp,
	struct MultiGPU plan[], int GPU_N
)
{
	int i;
	for(i=0;i<GPU_N;i++)
	{
		hipSetDevice(i);
		memset(plan[i].u0, 0, ntx*ntz*sizeof(float));
		memset(plan[i].u1, 0, ntx*ntz*sizeof(float));
		memset(plan[i].u2, 0, ntx*ntz*sizeof(float));
		memset(plan[i].seismogram_obs, 0, nt*rnmax*sizeof(float));
		memset(plan[i].seismogram_dir, 0, nt*rnmax*sizeof(float));
		memset(plan[i].seismogram_syn, 0, nt*rnmax*sizeof(float));
		memset(plan[i].seismogram_rms, 0, nt*rnmax*sizeof(float));
 		memset(plan[i].image_cor, 0, ntx*ntz*sizeof(float));
 		memset(plan[i].image_nor, 0, ntx*ntz*sizeof(float));
 		memset(plan[i].image_sources, 0, ntx*ntz*sizeof(float));
 		memset(plan[i].image_receivers, 0, ntx*ntz*sizeof(float));
	}
}

extern "C"
void getdevice(int *GPU_N)
{	
	hipGetDeviceCount(GPU_N);	
}
